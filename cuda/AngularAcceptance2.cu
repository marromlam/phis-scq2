#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//                                                                            //
//                       CUDA decay rate Bs -> mumuKK                         //
//                                                                            //
//  Created: 2019-01-25                                                       //
//                                                                            //
//                                                                            //
//                                                                            //
//                                                                            //
//                                                                            //
//                                                                            //
//                                                                            //
//                                                                            //
//                                                                            //
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// Inlude headers //////////////////////////////////////////////////////////////

#include <stdio.h>
#include <math.h>
// #include <thrust/complex.h>
#include <pycuda-complex.hpp>
//#include <hiprand.h>
//#include <hiprand/hiprand_kernel.h>
//#include "/scratch15/diego/gitcrap4/cuda/tag_gen.c"
//#include "/home3/marcos.romero/JpsiKKAna/cuda/somefunctions.c"
#include "/home3/marcos.romero/JpsiKKAna/cuda/Functions.c"
#define errf_const 1.12837916709551
#define xLim 5.33
#define yLim 4.29
__device__ double const sigma_threshold = 5.0;
#define time_acc_bins 40
#define spl_bins 7

extern "C"

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
{



__device__ double getNcoeffs(double APlon,
   double ASlon,
   double APpar,
   double APper,
   double CSP,
   int k)
{
  double nk;
  switch(k) {
  case 1:  nk = APlon*APlon;
   break;
  case 2:  nk = APpar*APpar;
   break;
  case 3:  nk = APper*APper;
   break;
  case 4:  nk = APper*APpar;
   break;
  case 5:  nk = APlon*APpar;
   break;
  case 6:  nk = APlon*APper;
   break;
  case 7:  nk = ASlon*ASlon;
   break;
  case 8:  nk = CSP*ASlon*APpar;
   break;
  case 9:  nk = CSP*ASlon*APper;
   break;
  case 10: nk = CSP*ASlon*APlon;
   break;
  default: printf("Wrong k index in nk, please check code %d\\n", k);
   return 0.;
  }
  return nk;
}

__device__ double getFcoeffs(double cosK,
   double cosL,
   double hphi,
   int k)
{
  double helsinthetaK = sqrt(1. - cosK*cosK);
  double helsinthetaL = sqrt(1. - cosL*cosL);
//     hphi -= M_PI;
//     double helsinphi = sin(-hphi);
//     double helcosphi = cos(-hphi);
  double helsinphi = sin(hphi);
  double helcosphi = cos(hphi);

  double fk;
  switch(k) {
  case 1:  fk = cosK*cosK*helsinthetaL*helsinthetaL;
   break;
  case 2:  fk = 0.5*helsinthetaK*helsinthetaK*(1.-helcosphi*helcosphi*helsinthetaL*helsinthetaL);
   break;
  case 3:  fk = 0.5*helsinthetaK*helsinthetaK*(1.-helsinphi*helsinphi*helsinthetaL*helsinthetaL);
   break;
  case 4:  fk = helsinthetaK*helsinthetaK*helsinthetaL*helsinthetaL*helsinphi*helcosphi;
   break;
  case 5:  fk = sqrt(2.)*helsinthetaK*cosK*helsinthetaL*cosL*helcosphi;
   break;
  case 6:  fk = -sqrt(2.)*helsinthetaK*cosK*helsinthetaL*cosL*helsinphi;
   break;
  case 7:  fk = helsinthetaL*helsinthetaL/3.;
   break;
  case 8:  fk = 2.*helsinthetaK*helsinthetaL*cosL*helcosphi/sqrt(6.);
   break;
  case 9:  fk = -2.*helsinthetaK*helsinthetaL*cosL*helsinphi/sqrt(6.);
   break;
  case 10: fk = 2.*cosK*helsinthetaL*helsinthetaL/sqrt(3.);
   break;
  default: printf("Wrong k index in fk, please check code %d\\n", k);
   return 0.;
  }
  return fk;
}

__device__ double getAcoeffs(double phisPlon,
   double phisSlon,
   double phisPpar,
   double phisPper,
   double deltaPlon,
   double deltaSlon,
   double deltaPpar,
   double deltaPper,
   double lambdaPlon,
   double lambdaSlon,
   double lambdaPpar,
   double lambdaPper,
   int k)
{
  double ak;
  switch(k) {
  case 1:  ak = 0.5*(1.+lambdaPlon*lambdaPlon);
   break;
  case 2:  ak = 0.5*(1.+lambdaPpar*lambdaPpar);
   break;
  case 3:  ak = 0.5*(1.+lambdaPper*lambdaPper);
   break;
  case 4:  ak = 0.5*(sin(deltaPper-deltaPpar) - lambdaPper*lambdaPpar*sin(deltaPper-deltaPpar-phisPper+phisPpar));
   break;
  case 5:  ak = 0.5*(cos(deltaPlon-deltaPpar) + lambdaPlon*lambdaPpar*cos(deltaPlon-deltaPpar-phisPlon+phisPpar));
   break;
  case 6:  ak = -0.5*(sin(deltaPlon-deltaPper) - lambdaPlon*lambdaPper*sin(deltaPlon-deltaPper-phisPlon+phisPper));
   break;
  case 7:  ak = 0.5*(1.+lambdaSlon*lambdaSlon);
   break;
  case 8:  ak = 0.5*(cos(deltaSlon-deltaPpar) - lambdaSlon*lambdaPpar*cos(deltaSlon-deltaPpar-phisSlon+phisPpar));
   break;
  case 9:  ak = -0.5*(sin(deltaSlon-deltaPper) + lambdaSlon*lambdaPper*sin(deltaSlon-deltaPper-phisSlon+phisPper));
   break;
  case 10: ak = 0.5*(cos(deltaSlon-deltaPlon) - lambdaSlon*lambdaPlon*cos(deltaSlon-deltaPlon-phisSlon+phisPlon));
   break;
  default: printf("Wrong k index in ak, please check code %d\\n", k);
   return 0.;
  }
  return ak;
}

__device__ double getBcoeffs(double phisPlon,
   double phisSlon,
   double phisPpar,
   double phisPper,
   double deltaPlon,
   double deltaSlon,
   double deltaPpar,
   double deltaPper,
   double lambdaPlon,
   double lambdaSlon,
   double lambdaPpar,
   double lambdaPper,
   int k)
{
  double bk;
  switch(k) {
  case 1:  bk = -lambdaPlon*cos(phisPlon);
   break;
  case 2:  bk = -lambdaPpar*cos(phisPpar);
   break;
  case 3:  bk = lambdaPper*cos(phisPper);
   break;
  case 4:  bk = 0.5*(lambdaPper*sin(deltaPper-deltaPpar-phisPper) + lambdaPpar*sin(deltaPpar-deltaPper-phisPpar));
   break;
  case 5:  bk = -0.5*(lambdaPlon*cos(deltaPlon-deltaPpar-phisPlon) + lambdaPpar*cos(deltaPpar-deltaPlon-phisPpar));
   break;
  case 6:  bk = 0.5*(lambdaPlon*sin(deltaPlon-deltaPper-phisPlon) + lambdaPper*sin(deltaPper-deltaPlon-phisPper));
   break;
  case 7:  bk = lambdaSlon*cos(phisSlon);
   break;
  case 8:  bk = 0.5*(lambdaSlon*cos(deltaSlon-deltaPpar-phisSlon) - lambdaPpar*cos(deltaPpar-deltaSlon-phisPpar));
   break;
  case 9:  bk = -0.5*(lambdaSlon*sin(deltaSlon-deltaPper-phisSlon) - lambdaPper*sin(deltaPper-deltaSlon-phisPper));
   break;
  case 10: bk = 0.5*(lambdaSlon*cos(deltaSlon-deltaPlon-phisSlon) - lambdaPlon*cos(deltaPlon-deltaSlon-phisPlon));
   break;
  default: printf("Wrong k index in bk, please check code %d\\n", k);
   return 0.;
  }
  return bk;
}

__device__ double getCcoeffs(double phisPlon,
   double phisSlon,
   double phisPpar,
   double phisPper,
   double deltaPlon,
   double deltaSlon,
   double deltaPpar,
   double deltaPper,
   double lambdaPlon,
   double lambdaSlon,
   double lambdaPpar,
   double lambdaPper,
   int k)
{

  double ck;
  switch(k) {
  case 1:  ck = 0.5*(1.-lambdaPlon*lambdaPlon);
   break;
  case 2:  ck = 0.5*(1.-lambdaPpar*lambdaPpar);
   break;
  case 3:  ck = 0.5*(1.-lambdaPper*lambdaPper);
   break;
  case 4:  ck = 0.5*(sin(deltaPper-deltaPpar) + lambdaPper*lambdaPpar*sin(deltaPper-deltaPpar-phisPper+phisPpar));
   break;
  case 5:  ck = 0.5*(cos(deltaPlon-deltaPpar) - lambdaPlon*lambdaPpar*cos(deltaPlon-deltaPpar-phisPlon+phisPpar));
   break;
  case 6:  ck = -0.5*(sin(deltaPlon-deltaPper) + lambdaPlon*lambdaPper*sin(deltaPlon-deltaPper-phisPlon+phisPper));
   break;
  case 7:  ck = 0.5*(1.-lambdaSlon*lambdaSlon);
   break;
  case 8:  ck = 0.5*(cos(deltaSlon-deltaPpar) + lambdaSlon*lambdaPpar*cos(deltaSlon-deltaPpar-phisSlon+phisPpar));
   break;
  case 9:  ck = -0.5*(sin(deltaSlon-deltaPper) - lambdaSlon*lambdaPper*sin(deltaSlon-deltaPper-phisSlon+phisPper));
   break;
  case 10: ck = 0.5*(cos(deltaSlon-deltaPlon) + lambdaSlon*lambdaPlon*cos(deltaSlon-deltaPlon-phisSlon+phisPlon));
   break;
  default: printf("Wrong k index in ck, please check code %d\\n", k);
   return 0.;
  }
  return ck;
}

__device__ double getDcoeffs(double phisPlon,
   double phisSlon,
   double phisPpar,
   double phisPper,
   double deltaPlon,
   double deltaSlon,
   double deltaPpar,
   double deltaPper,
   double lambdaPlon,
   double lambdaSlon,
   double lambdaPpar,
   double lambdaPper,
   int k)
{

  double dk;
  switch(k) {
  case 1:  dk = lambdaPlon*sin(phisPlon);
   break;
  case 2:  dk = lambdaPpar*sin(phisPpar);
   break;
  case 3:  dk = -lambdaPper*sin(phisPper);
   break;
  case 4:  dk = -0.5*(lambdaPper*cos(deltaPper-deltaPpar-phisPper) + lambdaPpar*cos(deltaPpar-deltaPper-phisPpar));
   break;
  case 5:  dk = -0.5*(lambdaPlon*sin(deltaPlon-deltaPpar-phisPlon) + lambdaPpar*sin(deltaPpar-deltaPlon-phisPpar));
   break;
  case 6:  dk = -0.5*(lambdaPlon*cos(deltaPlon-deltaPper-phisPlon) + lambdaPper*cos(deltaPper-deltaPlon-phisPper));
   break;
  case 7:  dk = -lambdaSlon*sin(phisSlon);
   break;
  case 8:  dk = 0.5*(lambdaSlon*sin(deltaSlon-deltaPpar-phisSlon) - lambdaPpar*sin(deltaPpar-deltaSlon-phisPpar));
   break;
  case 9:  dk = -0.5*(-lambdaSlon*cos(deltaSlon-deltaPper-phisSlon) + lambdaPper*cos(deltaPper-deltaSlon-phisPper));
   break;
  case 10: dk = 0.5*(lambdaSlon*sin(deltaSlon-deltaPlon-phisSlon) - lambdaPlon*sin(deltaPlon-deltaSlon-phisPlon));
   break;
  default: printf("Wrong k index in dk, please check code %d\\n", k);

   return 0.;
  }
  return dk;
}





__device__
pycuda::complex<double> getExponentialConvolution(double t, double gamma,
                                                  double omega, double sigma)
{
  pycuda::complex<double> I(0,1);
  pycuda::complex<double> z, fad;
  double sigma2 = sigma*sigma;

  if( t >sigma_threshold*sigma )
  {
    return  //2.*(sqrt(0.5*M_PI))* this was an old factor
  exp(-gamma*t+0.5*gamma*gamma*sigma2-0.5*omega*omega*sigma2)*
  (cos(omega*(t-gamma*sigma2)) + I*sin(omega*(t-gamma*sigma2)));
  }
  else
  {
    z   = (-I*(t-sigma2*gamma) - omega*sigma2)/(sigma*sqrt(2.));
    fad = faddeeva(z);
    fad = (pycuda::real(fad) - I*pycuda::imag(fad));
    return sqrt(0.5*M_PI)*exp(-0.5*t*t/sigma2)*fad;
  }
}


/*
__device__
pycuda::complex<double> calcM(double x, int n, double t, double sigma,
                              double gamma, double omega)
{
  pycuda::complex<double> conv_term;
  conv_term = getExponentialConvolution(t, gamma, omega, sigma)/(sqrt(0.5*M_PI));

  if (n == 0)
  {
    return pycuda::complex<double>(erf(x),0.)-conv_term;
  }
  else if (n == 1)
  {
    return 2.*(-pycuda::complex<double>(sqrt(1./M_PI)*exp(-x*x),0.)-x*conv_term);
  }
  else if (n == 2)
  {
    return 2.*(-2.*x*exp(-x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-(2.*x*x-1.)*conv_term);
  }
  else if (n == 3)
  {
    return 4.*(-(2.*x*x-1.)*exp(-x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-x*(2.*x*x-3.)*conv_term);
  }
  else if (n == 4)
  {
    return 4.*(exp(-x*x)*(6.*x+4.*x*x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-(3.-12.*x*x+4.*x*x*x*x)*conv_term);
  }
  else if (n == 5)
  {
    return 8.*(-(3.-12.*x*x+4.*x*x*x*x)*exp(-x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-x*(15.-20.*x*x+4.*x*x*x*x)*conv_term);
  }
  else if (n == 6)
  {
    return 8.*(-exp(-x*x)*(30.*x-40.*x*x*x+8.*x*x*x*x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-(-15.+90.*x*x-60.*x*x*x*x+8.*x*x*x*x*x*x)*conv_term);
  }
  return pycuda::complex<double>(0.,0.);
}




__device__
void intgTimeAcceptance(double time_terms[4], double sigma, double gamma,
                            double dgamma, double dm, double *knots,
                            double *coeffs, int n, double t0)
{
  // Add tUL to knots list
  knots[7] = 15; n += 1;
  int const N = 7+1;
  double x[N];

  double aux1 = 1./(sqrt(2.)*sigma);

  for(int i = 0; i < spl_bins+1; i++)
  {
    x[i] = (knots[i] - t0)*aux1;
  }

  // Fill S matrix                (TODO speed to be gained here - S is constant)
  double S[spl_bins][4][4];
  for (int bin=0; bin < spl_bins; ++bin)
  {
    for (int i=0; i<4; ++i)
    {
      for (int j=0; j<4; ++j)
      {
        if(i+j < 4)
        {
          S[bin][i][j] = getCoeff(coeffs,bin,i+j)
                         *Factorial(i+j)/Factorial(j)/Factorial(i)/pow(2.0,i+j);
        }
        else
        {
          S[bin][i][j] = 0.;
        }
      }
    }
  }


  pycuda::complex<double> z_sinh, K_sinh[4], M_sinh[spl_bins+1][4];
  pycuda::complex<double> z_cosh, K_cosh[4], M_cosh[spl_bins+1][4];
  pycuda::complex<double> z_trig, K_trig[4], M_trig[spl_bins+1][4];

  z_cosh = sigma*pycuda::complex<double>(gamma-0.5*dgamma,0.)/sqrt(2.);
  z_sinh = sigma*pycuda::complex<double>(gamma+0.5*dgamma,0.)/sqrt(2.);
  z_trig = sigma*pycuda::complex<double>(gamma,-dm)/sqrt(2.);

  // Fill Kn                 (only need to calculate this once per minimization)
  for (int j=0; j<4; ++j)
  {
    K_cosh[j] = Kn(z_cosh,j);
    K_sinh[j] = Kn(z_sinh,j);
    K_trig[j] = Kn(z_trig,j);
  }

  // Fill Mn
  for (int j=0; j<4; ++j)
  {
    for(int bin=0; bin < spl_bins+1; ++bin)
    {
      M_sinh[bin][j] = calcM(x[bin],j,knots[bin]-t0,sigma,gamma-0.5*dgamma,0.);
      M_cosh[bin][j] = calcM(x[bin],j,knots[bin]-t0,sigma,gamma+0.5*dgamma,0.);
      M_trig[bin][j] = calcM(x[bin],j,knots[bin]-t0,sigma,gamma,dm);
    }
  }

  // Fill the delta factors to multiply by the integrals
  double sigma_fact[4];
  for (int i=0; i<4; ++i)
  {
    sigma_fact[i] = pow(sigma*sqrt(2.), i+1)/sqrt(2.);
  }

  // Integral calculation for cosh, sinh, cos, sin terms
  double int_sinh = 0; double int_cosh = 0;
  pycuda::complex<double> int_trig = pycuda::complex<double>(0.,0.);

  for (int bin=0; bin < spl_bins; ++bin)
  {
    for (int j=0; j<=3; ++j)
    {
      for (int k=0; k<=3-j; ++k)
      {
        int_sinh += pycuda::real(S[bin][j][k]*(M_sinh[bin+1][j]-M_sinh[bin][j])
                    *K_cosh[k])*sigma_fact[j+k];

        int_cosh += pycuda::real(S[bin][j][k]*(M_cosh[bin+1][j]-M_cosh[bin][j])
                    *K_sinh[k])*sigma_fact[j+k];

        int_trig += S[bin][j][k]*(M_trig[bin+1][j] - M_trig[bin][j])
                    *K_trig[k]*sigma_fact[j+k];
      }
    }
  }

  // Fill itengral terms - 0:cosh, 1:sinh, 2:cos, 3:sin
  time_terms[0] = 0.5*(int_sinh + int_cosh);
  time_terms[1] = 0.5*(int_sinh - int_cosh);
  time_terms[2] = pycuda::real(int_trig);
  time_terms[3] = pycuda::imag(int_trig);

}



*/
/*
//This integral works for all decay times.
__device__
void integral4pitime_full_spline( double integral[2], double vNk[10], double vak[10],double vbk[10],
                                  double vck[10],double vdk[10], double *normweights, double G, double DG, double DM,
                                  double delta_t, double t_ll, double t_offset, int spline_Nknots, double *spline_knots, double *spline_coeffs)
{
    double time_terms[4] = {0., 0., 0., 0.};
    intgTimeAcceptance(time_terms, delta_t, G, DG, DM, spline_knots, spline_coeffs, spline_Nknots, t_offset) ;

    double int_ta = time_terms[0];
    double int_tb = time_terms[1];
    double int_tc = time_terms[2];
    double int_td = time_terms[3];

    for(int k=0; k<10; k++)
    {
        integral[0] += vNk[k]*normweights[k]*(vak[k]*int_ta + vbk[k]*int_tb + vck[k]*int_tc + vdk[k]*int_td);
        integral[1] += vNk[k]*normweights[k]*(vak[k]*int_ta + vbk[k]*int_tb - vck[k]*int_tc - vdk[k]*int_td);
    }
}
*/

__device__ double IntegralTimeA(double t_0, double t_1, double G,double DG)
{
    return (2*(DG*sinh(.5*DG*t_0) + 2*G*cosh(.5*DG*t_0))*exp(G*t_1) - 2*(DG*sinh(.5*DG*t_1) + 2*G*cosh(.5*DG*t_1))*exp(G*t_0))*exp(-G*(t_0 + t_1))/(-pow(DG, 2) + 4 *pow(G, 2));
}
__device__ double IntegralTimeB(double t_0, double t_1,double G,double DG)
{
    return (2*(DG*cosh(.5*DG*t_0) + 2*G*sinh(.5*DG*t_0))*exp(G*t_1) - 2*(DG*cosh(.5*DG*t_1) + 2*G*sinh(.5*DG*t_1))*exp(G*t_0))*exp(-G*(t_0 + t_1))/(-pow(DG, 2) + 4*pow(G, 2));
}
__device__ double IntegralTimeC(double t_0, double t_1,double G,double DM)
{
    return ((-DM*sin(DM*t_0) + G*cos(DM*t_0))*exp(G*t_1) + (DM*sin(DM*t_1) - G*cos(DM*t_1))*exp(G*t_0))*exp(-G*(t_0 + t_1))/(pow(DM, 2) + pow(G, 2));
}

__device__ double IntegralTimeD(double t_0, double t_1,double G,double DM)
{
    return ((DM*cos(DM*t_0) + G*sin(DM*t_0))*exp(G*t_1) - (DM*cos(DM*t_1) + G*sin(DM*t_1))*exp(G*t_0))*exp(-G*(t_0 + t_1))/(pow(DM, 2) + pow(G, 2));
}



__device__ void Integral4PiTime(double result[2],
                                double vnk[28],
                                double vak[28],
                                double vbk[28],
                                double vck[28],
                                double vdk[28],
                                double *normweights,
                                double Gamma, double DeltaGamma, double DeltaM,
                                double tLL, double tUL, double TimeOffset)
{
  double IntTimeA = IntegralTimeA(tLL, tUL, Gamma, DeltaGamma);
  double IntTimeB = IntegralTimeB(tLL, tUL, Gamma, DeltaGamma);
  double IntTimeC = IntegralTimeC(tLL, tUL, Gamma, DeltaM);
  double IntTimeD = IntegralTimeD(tLL, tUL, Gamma, DeltaM);

  for(int k=0; k<28 ; k++)
  {
    result[0] += vnk[k]*normweights[k]*(vak[k]*IntTimeA +
                                        vbk[k]*IntTimeB +
                                        vck[k]*IntTimeC +
                                        vdk[k]*IntTimeD);
    result[1] += vnk[k]*normweights[k]*(vak[k]*IntTimeA +
                                        vbk[k]*IntTimeB -
                                        vck[k]*IntTimeC -
                                        vdk[k]*IntTimeD);
  }
}




__device__
double getDiffRate(double *data, double G, double DG, double DM, double CSP,
                    double APlon, double ASlon, double APpar, double APper,
                    double phisPlon, double phisSlon, double phisPpar, double phisPper,
                    double deltaSlon, double deltaPlon, double deltaPpar, double deltaPper,
                    double lPlon, double lSlon, double lPpar, double lPper)
{
  // variables
  double cosK = data[0];
  double cosL = data[1];
  double hphi = data[2];
  double time = data[3];

  // double sigma_t 		= data[4];
  // double q_OS 			= data[5];
  // double qSlonSK 		= data[6];
  // double eta_OS 		= data[7];
  // double etaSlonSK 	= data[8];
  // int year 					= data[9];




/*
  double delta_t =  delta(sigma_t, sigma_t_a, sigma_t_b, sigma_t_c);

  double delta_t_1 = delta_1(sigma_t, fSlonigma_t, r_offset_pr, r_offsetSlonc, rSlonlope_pr, rSlonlopeSlonc, sigma_t_bar);
  double delta_t_2 = delta_2(sigma_t, fSlonigma_t, r_offset_pr, r_offsetSlonc, rSlonlope_pr, rSlonlopeSlonc, sigma_t_bar);

  double omega_OS = omega(eta_OS, p0_OS, dp0_OS, p1_OS, dp1_OS, p2_OS, dp2_OS, eta_bar_OS);
  double omega_bar_OS = omega_bar(eta_OS, p0_OS, dp0_OS, p1_OS, dp1_OS, p2_OS, dp2_OS, eta_bar_OS);
  double omegaSlonSK = omega(etaSlonSK, p0SlonSK, dp0SlonSK, p1SlonSK, dp1SlonSK, 0., 0., eta_barSlonSK);
  double omega_barSlonSK = omega_bar(etaSlonSK, p0SlonSK, dp0SlonSK, p1SlonSK, dp1SlonSK, 0., 0., eta_barSlonSK);

  double taggingPparrs_OS[3] = {omega_OS, omega_bar_OS, q_OS};
  double taggingPparrsSlonSK[3] = {omegaSlonSK, omega_barSlonSK, qSlonSK};

  fix_taggingPparrs(taggingPparrs_OS);
  fix_taggingPparrs(taggingPparrsSlonSK);

  omega_OS = taggingPparrs_OS[0];
  omega_bar_OS = taggingPparrs_OS[1];
  omegaSlonSK = taggingPparrsSlonSK[0];
  omega_barSlonSK = taggingPparrsSlonSK[1];

  if((taggingPparrs_OS[0] == 0.5 || taggingPparrs_OS[1] == 0.5) && (taggingPparrs_OS[0] != taggingPparrs_OS[1]))
  printf("OS tag mismatch!!! Check code %lf vs %lf and %lf \n", taggingPparrs_OS[0], taggingPparrs_OS[1], taggingPparrs_OS[2]);
  else
  q_OS = taggingPparrs_OS[2];

  if((taggingPparrsSlonSK[0] == 0.5 || taggingPparrsSlonSK[1] == 0.5) && (taggingPparrsSlonSK[0] != taggingPparrsSlonSK[1]))
  printf("SSK tag mismatch!!! Check code %lf vs %lf and %lf \n", taggingPparrsSlonSK[0], taggingPparrsSlonSK[1], taggingPparrsSlonSK[2]);
  else
  qSlonSK = taggingPparrsSlonSK[2];

*/




  // Time resolution -----------------------------------------------------------
  //     In order to remove the effects of conv, set delta_t = 0, so in this way
  //     you are running the first branch of getExponentialConvolution.
  pycuda::complex<double> exp_p, exp_m, exp_i;
  double t_offset = 0.0;//delta(sigma_t, sigma_t_mu_a, sigma_t_mu_b, sigma_t_mu_c);
  double delta_t  = 0.0;

  exp_p = getExponentialConvolution(time-t_offset, G + 0.5*DG, 0., delta_t);
  exp_m = getExponentialConvolution(time-t_offset, G - 0.5*DG, 0., delta_t);
  exp_i = getExponentialConvolution(time-t_offset,          G, DM, delta_t);

  double ta = pycuda::real(0.5*(exp_m + exp_p));     // cosh = (exp_m + exp_p)/2
  double tb = pycuda::real(0.5*(exp_m - exp_p));     // sinh = (exp_m - exp_p)/2
  double tc = pycuda::real(exp_i);                        // exp_i = cos + I*sin
  double td = pycuda::imag(exp_i);                        // exp_i = cos + I*sin
  //printf("%.8lf\t %.8lf\t %.8lf\t %.8lf\n", ta,tb,tc,td);



  // Flavor tagging ------------------------------------------------------------
  double omegaOSB = 0; double omegaOSBbar = 0;
  double omegaSSB = 0; double omegaSSBbar = 0;
  int tagOS = 0; int tagSS = 0;

  bool useTrueTag = 1;
  if (useTrueTag)
  {
    tagOS = 0.5;
  }

  //   tagOS = meas->tag_decision;
  //   tagSS = meas->tag_decision_ss;
  //
  //   double meas_omega    = meas->tag_omega;
  //   double meas_omega_ss = meas->tag_omega_ss;
  //
  //   double ma(0.99),mi(0);
  //
  //   omega_os_B =    std::max(std::min(params->tag_p0()    + params->tag_deltap0()/2.0    +(params->tag_p1()    + params->tag_deltap1()/2.0)    * (meas_omega - params->tag_eta()),ma),mi);
  //   omega_os_Bbar = std::max(std::min(params->tag_p0()    - params->tag_deltap0()/2.0    +(params->tag_p1()    - params->tag_deltap1()/2.0)    * (meas_omega - params->tag_eta()),ma),mi);
  //
  //   omega_ss_B =    std::max(std::min(params->tag_ss_p0() + params->tag_ss_deltap0()/2.0 +(params->tag_ss_p1() + params->tag_ss_deltap1()/2.0) * (meas_omega_ss - params->tag_ss_eta()),ma),mi);
  //   omega_ss_Bbar = std::max(std::min(params->tag_ss_p0() - params->tag_ss_deltap0()/2.0 +(params->tag_ss_p1() - params->tag_ss_deltap1()/2.0) * (meas_omega_ss - params->tag_ss_eta()),ma),mi);
  //
  //   if((1.0 + tagOS * (1-2*omega_os_B))*(1.0 + tagSS * (1-2*omega_ss_B)) == 0 && (1.0 - tagOS * (1-2*omega_os_Bbar))*(1.0 - tagSS * (1-2*omega_ss_Bbar)) == 0){
  //     omega_ss_Bbar=0.5;
  //     omega_ss_B=0.5;
  //     omega_os_Bbar=0.5;
  //     omega_os_B=0.5;
  //   }
  // }





















  // Decay-time acceptance -----------------------------------------------------
  //     To get rid of decay-time acceptance set dta to 1.0.
  double dta = 1.0;
  /*
  to be implemented
  */


  double vNk[10] = {0.,0.,0.,0.,0.,0.,0.,0.,0.,0.};
  double vak[10] = {0.,0.,0.,0.,0.,0.,0.,0.,0.,0.};
  double vbk[10] = {0.,0.,0.,0.,0.,0.,0.,0.,0.,0.};
  double vck[10] = {0.,0.,0.,0.,0.,0.,0.,0.,0.,0.};
  double vdk[10] = {0.,0.,0.,0.,0.,0.,0.,0.,0.,0.};

  double Nk, fk, ak, bk, ck, dk, hk_B, hk_Bbar;
  double pdfB = 0.0; double pdfBbar = 0.0;

  for(int k = 1; k <= 10; k++)
  {
    Nk = getNcoeffs(APlon,ASlon,APpar,APper,CSP,k);
    fk = 9./(16.*M_PI)*getFcoeffs(cosK,cosL,hphi,k);

    ak = getAcoeffs(phisPlon,phisSlon,phisPpar,phisPper,deltaPlon,deltaSlon,deltaPpar,deltaPper,lPlon,lSlon,lPpar,lPper,k);
    bk = getBcoeffs(phisPlon,phisSlon,phisPpar,phisPper,deltaPlon,deltaSlon,deltaPpar,deltaPper,lPlon,lSlon,lPpar,lPper,k);
    ck = getCcoeffs(phisPlon,phisSlon,phisPpar,phisPper,deltaPlon,deltaSlon,deltaPpar,deltaPper,lPlon,lSlon,lPpar,lPper,k);
    dk = getDcoeffs(phisPlon,phisSlon,phisPpar,phisPper,deltaPlon,deltaSlon,deltaPpar,deltaPper,lPlon,lSlon,lPpar,lPper,k);

    hk_B    = (ak*ta + bk*tb + ck*tc + dk*td);//old factor: 3./(4.*M_PI)*
    hk_Bbar = (ak*ta + bk*tb - ck*tc - dk*td);

    pdfB    += Nk*hk_B*fk;
    pdfBbar += Nk*hk_Bbar*fk;

    vNk[k-1] = 1.*Nk;
    vak[k-1] = 1.*ak; vbk[k-1] = 1.*bk; vck[k-1] = 1.*ck; vdk[k-1] = 1.*dk;
  }

  double normweights[10] = {1,1,1,0,0,0,1,0,0,0};
  double Int4PiTime[2] = {0.,0.};
  Integral4PiTime(Int4PiTime, vNk, vak, vbk, vck, vdk,
                  normweights,
                  G, DG, DM, 0.3, 15., 0.);
  double intB    = Int4PiTime[0];
  double intBbar = Int4PiTime[1];

  // Cooking the output --------------------------------------------------------
  double num = 1.0; double den = 1.0;
  num = dta*(
        (1+tagOS*(1-2*omegaOSB)   ) * (1+tagSS*(1-2*omegaSSB)   ) * pdfB +
        (1-tagOS*(1-2*omegaOSBbar)) * (1-tagSS*(1-2*omegaSSBbar)) * pdfBbar
        );
  den = 1.0*(
        (1+tagOS*(1-2*omegaOSB)   ) * (1+tagSS*(1-2*omegaSSB)   ) * intB +
        (1-tagOS*(1-2*omegaOSBbar)) * (1-tagSS*(1-2*omegaSSBbar)) * intBbar
        );
  printf("t=%+lf\tcosK=%+lf\tcosL=%+lf\thphi=%+lf\tpdf=%+lf\tipdf=%+lf\t --> pdf/ipdf=%+lf\n", time,cosK,cosL,hphi, num,den,num/den);
  return num;///den;
}



__device__
void getAngularWeights(double *dtrue, double *dreco, double *w10,
                       double G, double DG, double DM, double CSP,
                       double APlon, double ASlon, double APpar, double APper,
                       double phisPlon, double phisSlon, double phisPpar,
                       double phisPper, double deltaSlon, double deltaPlon,
                       double deltaPpar, double deltaPper, double lPlon,
                       double lSlon, double lPpar, double lPper)
{
  double fk = 0.0;
  double pdf_reco = getDiffRate(dreco,  G,  DG,  DM,  CSP,
                                APlon,  ASlon,  APpar,  APper,
                                phisPlon,  phisSlon,  phisPpar,  phisPper,
                                deltaSlon,  deltaPlon,  deltaPpar,  deltaPper,
                                lPlon,  lSlon,  lPpar,  lPper);

  for(int k = 0; k < 10; k++)
  {
    fk     = getFcoeffs(dtrue[0],dtrue[1],dtrue[2],k+1);
    w10[k] = 9./(16.*M_PI)*fk/pdf_reco;
  }
}



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::pyDiffRate //////////////////////////////////////////////////////////

__global__
void pyDiffRate(double *data, double *lkhd,
                double G, double DG, double DM, double CSP, double APlon,
                double ASlon, double APpar, double APper, double phisPlon,
                double phisSlon, double phisPpar, double phisPper,
                double deltaSlon, double deltaPlon, double deltaPpar,
                double deltaPper, double lPlon, double lSlon, double lPpar,
                double lPper,
                int Nevt)
{
  int row = threadIdx.x + blockDim.x * blockIdx.x;
  if (row >= Nevt) { return; }
  //printf("%lf\n", data[row]);

  double data4[4] = {data[row*4+0],data[row*4+1],data[row*4+2],data[row*4+3]};

  lkhd[row] = getDiffRate(data4,
                          G, DG, DM, CSP, APlon, ASlon, APpar, APper, phisPlon,
                          phisSlon, phisPpar, phisPper, deltaSlon, deltaPlon,
                          deltaPpar, deltaPper, lPlon, lSlon, lPpar, lPper);

}

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::pyFcoeffs ///////////////////////////////////////////////////////////

__global__
void pyFcoeffs(double *data, double *fk,  int Nevt)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int k = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= Nevt) { return; }
  fk[i*10+k]= 9./(16.*M_PI)*getFcoeffs(data[i*4+0],data[i*4+1],data[i*4+2],k+1);
}

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::getAngularWeights ///////////////////////////////////////////////////

__global__
void pyAngularWeights(double *dtrue, double *dreco, double *w,
                      double G, double DG, double DM, double CSP,
                      double APlon, double ASlon, double APpar, double APper,
                      double phisPlon, double phisSlon, double phisPpar,
                      double phisPper, double deltaSlon, double deltaPlon,
                      double deltaPpar, double deltaPper, double lPlon,
                      double lSlon, double lPpar, double lPper,
                      int Nevt)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x; // loop over events
  if (i >= Nevt) { return; }
  double w10[10]     = {0,0,0,0,0,0,0,0,0,0};
  double vec_true[4] = {dtrue[i*4+0],dtrue[i*4+1],dtrue[i*4+2],dtrue[i*4+3]};
  double vec_reco[4] = {dreco[i*4+0],dreco[i*4+1],dreco[i*4+2],dreco[i*4+3]};
  getAngularWeights(vec_true, vec_reco, w10,
                    G, DG, DM, CSP, APlon, ASlon, APpar, APper, phisPlon,
                    phisSlon, phisPpar, phisPper, deltaSlon, deltaPlon,
                    deltaPpar, deltaPper, lPlon, lSlon, lPpar, lPper);
  for(int k = 0; k < 10; k++)
  {
    w[i*10+k] = w10[k];
  }
}

////////////////////////////////////////////////////////////////////////////////






}
