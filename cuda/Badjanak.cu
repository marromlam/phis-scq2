#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//                                                                            //
//                      OPENCL decay rate Bs -> mumuKK                        //
//                                                                            //
//   Created: 2019-11-18                                                      //
//  Modified: 2019-11-21                                                      //
//    Author: Marcos Romero                                                   //
//                                                                            //
//    This file is part of p-scq packages, Santiago's framework for the       //
//                     phi_s analysis in Bs -> Jpsi K+ K-                     //
//                                                                            //
//  This file contains the following __global__s:                             //
//    * pyDiffRate: Computes Bs2MuMuKK pdf looping over the events. Now it    //
//                  handles a binned X_M fit without splitting beforehand the //
//                  data --it launches a thread per mass bin.                 //
//                                                                            //
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// Include headers /////////////////////////////////////////////////////////////

#include <stdio.h>
#include <math.h>
#include <pycuda-complex.hpp>

// Debugging 0 [0,1,2,3,>3]
#define DEBUG {DEBUG}
#define DEBUG_EVT {DEBUG_EVT}

// Flags
#define USE_TIME_ACC {USE_TIME_ACC}
#define USE_TIME_OFFSET {USE_TIME_OFFSET}
#define USE_TIME_RES {USE_TIME_RES}
#define USE_PERFTAG {USE_PERFTAG}
#define USE_TRUETAG {USE_TRUETAG}

// Time resolution parameters
#define SIGMA_T {SIGMA_T}

// Time acceptance parameters
#define NKNOTS {NKNOTS}
#define NTIMEBINS {NTIMEBINS}
__device__ double const KNOTS[NKNOTS] = {KNOTS};

__device__ double const SIGMA_THRESHOLD = 5.0;
__device__ int const TIME_ACC_BINS = 40;
__device__ int const SPL_BINS = 7;


// PDF parameters
#define NMASSBINS {NMASSBINS}
__device__ double const X_M[8] = {X_M};
__device__ double const TRISTAN[10] = {TRISTAN};

// Include disciplines
//     They follow the next tree, which means that its only necessay to include
//     AngularAcceptance.cu in order to load all of them.
//         AngularAcceptance
//           |– DifferentialCrossRate
//               |- DecayTimeAcceptance
//                   |– Functions
//               |– TimeAngularDistribution
#include "AngularAcceptance.cu"

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::pyDiffRate //////////////////////////////////////////////////////////

__global__
void pyDiffRate(double *data, double *lkhd,
                double G, double DG, double DM,
                const double * CSP,
                const double * ASlon,
                const double * APlon,
                const double * APpar,
                const double * APper,
                double pSlon, double pPlon, double pPpar, double pPper,
                const double * deltaSlon,
                double deltaPlon, double deltaPpar, double deltaPper,
                double lSlon, double lPlon, double lPpar, double lPper,
                double tLL, double tUL,
                double *coeffs,
                int Nevt)
{{
  int evt = threadIdx.x + blockDim.x * blockIdx.x;
  int bin = threadIdx.y + blockDim.y * blockIdx.y;
  if (evt >= Nevt) {{ return; }}

  double shit[28];                                // check why this is mandatory
  for (int index =0; index < 28; index++)
  {{
    shit[index] = coeffs[index];
  }}

  double mass = data[evt*7+4];
  //printf("mass=%+lf",mass);
  double data4[6] = {{data[evt*7+0], // cosK
                      data[evt*7+1], // cosL
                      data[evt*7+2], // hphi
                      data[evt*7+3], // time
                      data[evt*7+5], // sigma_t
                      data[evt*7+6]  // flavour
                    }};

  if (blockDim.y > 1)                   // if fitting binned X_M spectrum
  {{
    if ((mass >= X_M[bin]) && (mass < X_M[bin+1]))
    {{
      lkhd[evt] = getDiffRate(data4,
                              G, DG, DM, CSP[bin],
                              APlon[bin], ASlon[bin], APpar[bin], APper[bin],
                              pPlon, pSlon, pPpar, pPper,
                              deltaSlon[bin], deltaPlon, deltaPpar, deltaPper,
                              lPlon, lSlon, lPpar, lPper,
                              tLL, tUL,
                              shit, 1);
    }}
  }}
  else
  {{
    lkhd[evt] = getDiffRate(data4,
                            G, DG, DM, CSP[0],
                            ASlon[0], APlon[0], APpar[0], APper[0],
                            pPlon,    pSlon,    pPpar, pPper,
                            deltaSlon[0], deltaPlon, deltaPpar, deltaPper,
                            lPlon, lSlon, lPpar, lPper,
                            tLL, tUL,
                            shit, 1);
  }}


}}

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::pyFcoeffs ///////////////////////////////////////////////////////////

__global__
void pyFcoeffs(double *data, double *fk,  int Nevt)
{{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int k = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= Nevt) {{ return; }}
  fk[i*10+k]= 9./(16.*M_PI)*getF(data[i*4+0],data[i*4+1],data[i*4+2],k+1);
}}

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::getAngularWeights ///////////////////////////////////////////////////

__global__
void pyAngularWeights(double *data, double *w,
                      double G, double DG, double DM, double CSP,
                      double ASlon, double APlon, double APpar, double APper,
                      double pSlon, double pPlon, double pPpar, double pPper,
                      double dSlon, double dPlon, double dPpar, double dPper,
                      double lSlon, double lPlon, double lPpar, double lPper,
                      double tLL, double tUL,
                      double *coeffs,
                      int Nevt)
{{
  int evt = threadIdx.x + blockDim.x * blockIdx.x;
  if (evt >= Nevt) {{ return; }}

  double w10[10]     = {{0,0,0,0,0,0,0,0,0,0}};
  double vec_true[6] = {{data[evt*4+0], // cosK
                         data[evt*4+1], // cosL
                         data[evt*4+2], // hphi
                         data[evt*4+3], // time
                         0,              // sigma_t
                         0               // flavour
                       }};

  getAngularWeights(vec_true, w10,
                    G, DG, DM, CSP,
                    ASlon, APlon, APpar, APper,
                    pSlon, pPlon, pPpar, pPper,
                    dSlon, dPlon, dPpar, dPper,
                    lSlon, lPlon, lPpar, lPper,
                    tLL, tUL,
                    coeffs);

  __syncthreads();
  for(int k = 0; k < 10; k++)
  {{
    atomicAdd( &w[0]+k , w10[k]);
  }}
  //__syncthreads();

}}

////////////////////////////////////////////////////////////////////////////////
