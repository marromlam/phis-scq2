#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//                                                                            //
//                      OPENCL decay-time acceptance                          //
//                                                                            //
//   Created: 2019-01-28                                                      //
//  Modified: 2019-11-27                                                      //
//    Author: Marcos Romero                                                   //
//                                                                            //
//    This file is part of phis-scq packages, Santiago's framework for the    //
//                     phi_s analysis in Bs -> Jpsi K+ K-                     //
//                                                                            //
//  This file contains the following __kernels:                               //
//    * [none]                                                                //
//                                                                            //
//  TODO: The way complex numbers are handled is a bit neolithic, but as      //
//        far as openCL does not provide a standard library, this is the      //
//        only solution avaliable                                             //
//  TODO: Finish openCL translation of decay-time acceptance getM and getK    //
//        functions, that are incomplete por dta analysis but right for       //
//        computing standard pdf                                              //
//                                                                            //
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// Include headers /////////////////////////////////////////////////////////////

#include <stdio.h>
#include <math.h>
#include <pycuda-complex.hpp>

#include "Functions.cu"

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// Functions ///////////////////////////////////////////////////////////////////



__device__
int getTimeBin(double t)
{
  int _i = 0;
  int _n = NKNOTS-1;
  //printf("%d\n", _n);
  while(_i <= _n )
  {
    if( t < KNOTS[_i] ) {break;}
    _i++;
  }
  if (0 == _i) {printf("WARNING: t=%lf below first knot!\n",t);}
  return _i - 1;

}



__device__
double getKnot(int i)
{
  if (i<=0) {
    i = 0;
  }
  else if (i>=NKNOTS) {
    i = NKNOTS;
  }
  return KNOTS[i];
}



__device__
double getCoeff(double *mat, int r, int c)
{
  return mat[4*r+c];
}




__device__
double calcTimeAcceptance(double t, double *coeffs, double tLL, double tUL)
{
  int bin   = getTimeBin(t);
  if (t < tLL) { return 0.0; }
  if (t > tUL) { return 0.0; }

  double c0 = getCoeff(coeffs,bin,0);
  double c1 = getCoeff(coeffs,bin,1);
  double c2 = getCoeff(coeffs,bin,2);
  double c3 = getCoeff(coeffs,bin,3);

  double result = (c0 + t*(c1 + t*(c2 + t*c3)));
  if (DEBUG >= 3 && ( threadIdx.x + blockDim.x * blockIdx.x == 0))
  {
    printf("TIME ACC  : t=%lf\tbin=%d\tc=[%+lf\t%+lf\t%+lf\t%+lf]\tdta=%lf\n",
           t,bin,c0,c1,c2,c3,result);
  }

  return result;
}



__device__
pycuda::complex<double> getExponentialConvolution(double t, double G, double omega, double sigma)
{

  double sigma2 = sigma*sigma;
  pycuda::complex<double> I(0,1);

  if( t >SIGMA_THRESHOLD*sigma )
  {//2.*(sqrt(0.5*M_PI))* this was an old factor
    return exp(-G*t+0.5*G*G*sigma2-0.5*omega*omega*sigma2)*(cos(omega*(t-G*sigma2)) + I*sin(omega*(t-G*sigma2)));
  }
  else
  {
    pycuda::complex<double> z, fad, result;
    z   = (-I*(t-sigma2*G) - omega*sigma2)/(sigma*sqrt(2.));
    fad = faddeeva(z);
    fad = (pycuda::real(fad) - I*pycuda::imag(fad));
    return sqrt(0.5*M_PI)*exp(-0.5*t*t/sigma2)*fad;
  }
}






































__device__
pycuda::complex<double> getK(pycuda::complex<double> z, int n)
{
  if (n == 0)
  {
    return 1./(2.*z);
  }
  else if (n == 1)
  {
    return 1./(2.*z*z);
  }
  else if (n == 2)
  {
    return 1./z*(1.+1./(z*z));
  }
  else if (n == 3)
  {
    return 3./(z*z)*(1.+1./(z*z));
  }
  else if (n == 4)
  {
    return 6./z*(1.+2./(z*z)+2./(z*z*z*z));
  }
  else if (n == 5)
  {
    return 30./(z*z)*(1.+2./(z*z)+2./(z*z*z*z));
  }
  else if (n == 6)
  {
    return 60./z*(1.+3./(z*z)+6./(z*z*z*z)+6./(z*z*z*z*z*z));
  }

  return pycuda::complex<double>(0.,0.);

}















__device__
pycuda::complex<double> getM(double x, int n, double t, double sigma,
                              double gamma, double omega)
{
  pycuda::complex<double> conv_term;
  conv_term = getExponentialConvolution(t,gamma,omega,sigma)/(sqrt(0.5*M_PI));

  if (n == 0)
  {
    return pycuda::complex<double>(erf(x),0.)-conv_term;
  }
  else if (n == 1)
  {
    return 2.*(-pycuda::complex<double>(sqrt(1./M_PI)*exp(-x*x),0.)-x*conv_term);
  }
  else if (n == 2)
  {
    return 2.*(-2.*x*exp(-x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-(2.*x*x-1.)*conv_term);
  }
  else if (n == 3)
  {
    return 4.*(-(2.*x*x-1.)*exp(-x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-x*(2.*x*x-3.)*conv_term);
  }
  else if (n == 4)
  {
    return 4.*(exp(-x*x)*(6.*x+4.*x*x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-(3.-12.*x*x+4.*x*x*x*x)*conv_term);
  }
  else if (n == 5)
  {
    return 8.*(-(3.-12.*x*x+4.*x*x*x*x)*exp(-x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-x*(15.-20.*x*x+4.*x*x*x*x)*conv_term);
  }
  else if (n == 6)
  {
    return 8.*(-exp(-x*x)*(30.*x-40.*x*x*x+8.*x*x*x*x*x)*pycuda::complex<double>(sqrt(1./M_PI),0.)-(-15.+90.*x*x-60.*x*x*x*x+8.*x*x*x*x*x*x)*conv_term);
  }
  return pycuda::complex<double>(0.,0.);
}




__device__
void intgTimeAcceptance(double time_terms[4], double sigma,
                        double G, double DG, double DM,
                        double *knots, double *coeffs, int n, double t0)
{
  // Add tUL to knots list
  knots[NTIMEBINS] = 15; n += 1;
  double x[NTIMEBINS];

  double aux1 = 1./(sqrt(2.0)*sigma);
  pycuda::complex<double> aux2 = (sigma/(sqrt(2.0)),0);

  for(int i = 0; i < SPL_BINS+1; i++)
  {
    x[i] = (knots[i] - t0)*aux1;
  }

  // Fill S matrix                (TODO speed to be gained here - S is constant)
  double S[SPL_BINS][4][4];
  for (int bin=0; bin < SPL_BINS; ++bin)
  {
    for (int i=0; i<4; ++i)
    {
      for (int j=0; j<4; ++j)
      {
        if(i+j < 4)
        {
          S[bin][i][j] = getCoeff(coeffs,bin,i+j)
                         *factorial(i+j)/factorial(j)/factorial(i)/pow(2.0,i+j);
        }
        else
        {
          S[bin][i][j] = 0.;
        }
      }
    }
  }

  pycuda::complex<double> z_sinh, K_sinh[4], M_sinh[SPL_BINS+1][4];
  pycuda::complex<double> z_cosh, K_cosh[4], M_cosh[SPL_BINS+1][4];
  pycuda::complex<double> z_trig, K_trig[4], M_trig[SPL_BINS+1][4];

  z_cosh = aux2 * pycuda::complex<double>(G-0.5*DG,  0);
  z_sinh = aux2 * pycuda::complex<double>(G+0.5*DG,  0);
  z_trig = aux2 * pycuda::complex<double>(       G,-DM);

  // Fill Kn                 (only need to calculate this once per minimization)
  for (int j=0; j<4; ++j)
  {
    K_cosh[j] = getK(z_cosh,j);
    K_sinh[j] = getK(z_sinh,j);
    K_trig[j] = getK(z_trig,j);
  }

  // Fill Mn
  for (int j=0; j<4; ++j)
  {
    for(int bin=0; bin < SPL_BINS+1; ++bin)
    {
      M_sinh[bin][j] = getM(x[bin],j,knots[bin]-t0,sigma,G-0.5*DG,0.);
      M_cosh[bin][j] = getM(x[bin],j,knots[bin]-t0,sigma,G+0.5*DG,0.);
      M_trig[bin][j] = getM(x[bin],j,knots[bin]-t0,sigma,G,DM);
    }
  }

  // Fill the delta factors to multiply by the integrals
  double sigma_fact[4];
  for (int i=0; i<4; ++i)
  {
    sigma_fact[i] = pow(sigma*sqrt(2.), i+1)/sqrt(2.);
  }

  // Integral calculation for cosh, sinh, cos, sin terms
  double int_sinh = 0;
  double int_cosh = 0;
  pycuda::complex<double> int_trig = pycuda::complex<double>(0.,0.);

  for (int ibin=0; ibin < SPL_BINS; ++ibin)
  {
    for (int j=0; j<=3; ++j)
    {
      for (int k=0; k<=3-j; ++k)
      {
        int_sinh += pycuda::real(S[ibin][j][k]*
                                 (M_sinh[ibin+1][j] - M_sinh[ibin][j])*
                                 K_cosh[k])*
                                 sigma_fact[j+k];

        int_cosh += pycuda::real(S[ibin][j][k]*
                                 (M_cosh[ibin+1][j] - M_cosh[ibin][j])*
                                 K_sinh[k])*
                                 sigma_fact[j+k];

        int_trig +=  S[ibin][j][k]*
                     (M_trig[ibin+1][j] - M_trig[ibin][j])*
                     K_trig[k]*
                     sigma_fact[j+k];
      }
    }
  }

  // Fill itengral terms - 0:cosh, 1:sinh, 2:cos, 3:sin
  time_terms[0] = 0.5*(int_sinh + int_cosh);
  time_terms[1] = 0.5*(int_sinh - int_cosh);
  time_terms[2] = pycuda::real(int_trig);
  time_terms[3] = pycuda::imag(int_trig);

  if (DEBUG > 3 && ( threadIdx.x + blockDim.x * blockIdx.x == 0) )
  {
    printf("INTEGRAL           : ta=%.8lf\ttb=%.8lf\ttc=%.8lf\ttd=%.8lf\n",
           time_terms[0],time_terms[1],time_terms[2],time_terms[3]);
  }
}



__device__
void integralFullSpline( double result[2],
                         double vn[10], double va[10],double vb[10], double vc[10],double vd[10],
                         double *norm, double G, double DG, double DM,
                         double delta_t,
                         double t_ll,
                         double t_offset,
                         double nknots, double *knots,
                         double *spline_coeffs)
{
  double integrals[4] = {0., 0., 0., 0.};
  intgTimeAcceptance(integrals, delta_t, G, DG, DM,
                     knots, spline_coeffs, nknots, t_offset) ;
  double ta = integrals[0];
  double tb = integrals[1];
  double tc = integrals[2];
  double td = integrals[3];

  for(int k=0; k<10; k++)
  {
    result[0] += vn[k]*norm[k]*(va[k]*ta + vb[k]*tb + vc[k]*tc + vd[k]*td);
    result[1] += vn[k]*norm[k]*(va[k]*ta + vb[k]*tb - vc[k]*tc - vd[k]*td);
  }
}



/*
__device__
pycuda::complex<double> getExponentialConvolution(double t, double gamma,
                                                  double omega, double sigma)
{
  pycuda::complex<double> I(0,1);
  pycuda::complex<double> z, fad;
  double sigma2 = sigma*sigma;

  if( t >sigma_threshold*sigma )
  {
    return  //2.*(sqrt(0.5*M_PI))* this was an old factor
  exp(-gamma*t+0.5*gamma*gamma*sigma2-0.5*omega*omega*sigma2)*
  (cos(omega*(t-gamma*sigma2)) + I*sin(omega*(t-gamma*sigma2)));
  }
  else
  {
    z   = (-I*(t-sigma2*gamma) - omega*sigma2)/(sigma*sqrt(2.));
    fad = faddeeva(z);
    fad = (pycuda::real(fad) - I*pycuda::imag(fad));
    return sqrt(0.5*M_PI)*exp(-0.5*t*t/sigma2)*fad;
  }
}
*/



////////////////////////////////////////////////////////////////////////////////
