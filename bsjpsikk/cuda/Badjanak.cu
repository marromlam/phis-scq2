#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//                                                                            //
//                      OPENCL decay rate Bs -> mumuKK                        //
//                                                                            //
//   Created: 2019-11-18                                                      //
//  Modified: 2019-11-21                                                      //
//    Author: Marcos Romero                                                   //
//                                                                            //
//    This file is part of phis-scq packages, Santiago's framework for the    //
//                     phi_s analysis in Bs -> Jpsi K+ K-                     //
//                                                                            //
//  This file contains the following __global__s:                             //
//    * pyDiffRate: Computes Bs2MuMuKK pdf looping over the events. Now it    //
//                  handles a binned X_M fit without splitting beforehand the //
//                  data --it launches a thread per mass bin.                 //
//                                                                            //
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// Include headers /////////////////////////////////////////////////////////////

#include <stdio.h>
#include <math.h>
#include <pycuda-complex.hpp>

// Debugging 0 [0,1,2,3,>3]
#define DEBUG {DEBUG}
#define DEBUG_EVT {DEBUG_EVT}

// Flags
#define USE_TIME_ACC {USE_TIME_ACC}
#define USE_TIME_OFFSET {USE_TIME_OFFSET}
#define USE_TIME_RES {USE_TIME_RES}
#define USE_PERFTAG {USE_PERFTAG}
#define USE_TRUETAG {USE_TRUETAG}

// Time resolution parameters
#define SIGMA_T {SIGMA_T}

// Time acceptance parameters
#define NKNOTS {NKNOTS}
#define NTERMS {NTERMS}
#define NTIMEBINS {NTIMEBINS}
__device__ double KNOTS[NKNOTS] = {KNOTS};
__device__ double ANG_ACC[NTERMS] = {ANG_ACC};


__device__ double const SIGMA_THRESHOLD = 5.0;
//__device__ int const TIME_ACC_BINS = 40;
__device__ int const SPL_BINS = 7;


// PDF parameters
#define NMASSBINS {NMASSBINS}
__device__ double const X_M[7] = {X_M};
//__device__ double const TRISTAN[10] = {TRISTAN};

// Include disciplines
//     They follow the next tree, which means that its only necessay to include
//     AngularAcceptance.cu in order to load all of them.
//         AngularAcceptance
//           |– DifferentialCrossRate
//               |- DecayTimeAcceptance
//                   |– Functions
//               |– TimeAngularDistribution
#include "AngularAcceptance.cu"
//#include "DifferentialCrossRate.cu"

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::pyDiffRate //////////////////////////////////////////////////////////

__global__
void pyDiffRate(double *data, double *lkhd,
                // Time-dependent angular distribution
                double G, double DG, double DM,
                double * CSP,
                double *ASlon, double *APlon, double *APpar, double *APper,
                double  pSlon, double  pPlon, double  pPpar, double  pPper,
                double *dSlon, double  dPlon, double  dPpar, double  dPper,
                double  lSlon, double  lPlon, double  lPpar, double  lPper,
                // Time limits
                double tLL, double tUL,
                // Time resolution
                double sigma_offset, double sigma_slope, double sigma_curvature,
                double mu,
                // Flavor tagging
                double eta_bar_os, double eta_bar_ss,
                double p0_os,  double p1_os, double p2_os,
                double p0_ss,  double p1_ss, double p2_ss,
                double dp0_os, double dp1_os, double dp2_os,
                double dp0_ss, double dp1_ss, double dp2_ss,
                // Time acceptance
                double *coeffs,
                // Angular acceptance
                double *angular_weights, int use_fk, int bins,
                int Nevt)
{{
  int evt = threadIdx.x + blockDim.x * blockIdx.x;
  //int bin = threadIdx.y + blockDim.y * blockIdx.y;
  if (evt >= Nevt) {{ return; }}

  double mass = data[evt*10+4];
  double data4[9] = {{data[evt*10+0], // cosK
                      data[evt*10+1], // cosL
                      data[evt*10+2], // hphi
                      data[evt*10+3], // time
                      data[evt*10+5], // sigma_t
                      data[evt*10+6], // qOS
                      data[evt*10+7], // qSS
                      data[evt*10+8], // etaOS
                      data[evt*10+9]  // etaSS
                    }};




  if (bins>1)
  {{
  for (int bin =0; bin<bins; bin++)
  {{
    if ((mass >= X_M[bin]) && (mass < X_M[bin+1]))
    {{
      lkhd[evt] = getDiffRate(data4,
                              G, DG, DM, CSP[bin],
                              ASlon[bin], APlon[bin], APpar[bin], APper[bin],
                              pSlon,      pPlon,      pPpar,      pPper,
                              dSlon[bin], dPlon,      dPpar,      dPper,
                              lSlon,      lPlon,      lPpar,      lPper,
                              tLL, tUL,
                              sigma_offset, sigma_slope, sigma_curvature,
                              mu,
                              eta_bar_os, eta_bar_ss,
                              p0_os,  p1_os, p2_os,
                              p0_ss,  p1_ss, p2_ss,
                              dp0_os, dp1_os, dp2_os,
                              dp0_ss, dp1_ss, dp2_ss,
                              coeffs,
                              angular_weights, use_fk);
    }}
  }}
  }}
  else
  {{
    lkhd[evt] = getDiffRate(data4,
                            G, DG, DM, CSP[0],
                            ASlon[0], APlon[0], APpar[0], APper[0],
                            pSlon,    pPlon,    pPpar,    pPper,
                            dSlon[0], dPlon,    dPpar,    dPper,
                            lSlon,    lPlon,    lPpar,    lPper,
                            tLL, tUL,
                            sigma_offset, sigma_slope, sigma_curvature,
                            mu,
                            eta_bar_os, eta_bar_ss,
                            p0_os,  p1_os, p2_os,
                            p0_ss,  p1_ss, p2_ss,
                            dp0_os, dp1_os, dp2_os,
                            dp0_ss, dp1_ss, dp2_ss,
                            coeffs,
                            angular_weights, use_fk);
  }}


}}

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::pyFcoeffs ///////////////////////////////////////////////////////////

__global__
void pyFcoeffs(double *data, double *fk,  int Nevt)
{{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int k = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= Nevt) {{ return; }}
  fk[i*10+k]= 9./(16.*M_PI)*getF(data[i*10+0],data[i*10+1],data[i*10+2],k+1);
}}

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::getAngularWeights ///////////////////////////////////////////////////

__global__
void pyAngularWeights(double *dtrue, double *dreco, double *weight, double *w,
                      // Time-dependent angular distribution
                      double G, double DG, double DM,
                      double * CSP,
                      double *ASlon, double *APlon, double *APpar, double *APper,
                      double  pSlon, double  pPlon, double  pPpar, double  pPper,
                      double *dSlon, double  dPlon, double  dPpar, double  dPper,
                      double  lSlon, double  lPlon, double  lPpar, double  lPper,
                      double tLL, double tUL,
                      double sigma_offset, double sigma_slope, double sigma_curvature,
                      double mu,
                      // Flavor tagging
                      double eta_bar_os, double eta_bar_ss,
                      double p0_os,  double p1_os, double p2_os,
                      double p0_ss,  double p1_ss, double p2_ss,
                      double dp0_os, double dp1_os, double dp2_os,
                      double dp0_ss, double dp1_ss, double dp2_ss,
                      // Time acceptance
                      int nknots, double *knots, double *coeffs,
                      // Angular acceptance
                      double *angular_weights,
                      int Nevt)
{{
  int evt = threadIdx.x + blockDim.x * blockIdx.x;
  if (evt >= Nevt) {{ return; }}

  double w10[10]     = {{0,0,0,0,0,0,0,0,0,0}};
  double vec_true[9] = {{dtrue[evt*10+0], // cosK
                         dtrue[evt*10+1], // cosL
                         dtrue[evt*10+2], // hphi
                         dtrue[evt*10+3], // time
                         dtrue[evt*10+5], // sigma_t
                         dtrue[evt*10+6], // qOS
                         dtrue[evt*10+6], // qSS
                         0,              // etaOS
                         0               // etaSS
                       }};
  double vec_reco[9] = {{dreco[evt*10+0], // cosK
                         dreco[evt*10+1], // cosL
                         dreco[evt*10+2], // hphi
                         dreco[evt*10+3], // time
                         dreco[evt*10+5], // sigma_t
                         dreco[evt*10+6], // qOS
                         dreco[evt*10+6], // qSS
                         0,              // etaOS
                         0               // etaSS
                       }};

  getAngularWeights(vec_true, vec_reco, weight[evt], w10,
                    G, DG, DM, CSP[0],
                    ASlon[0], APlon[0], APpar[0], APper[0],
                    pSlon,    pPlon,    pPpar,    pPper,
                    dSlon[0], dPlon,    dPpar,    dPper,
                    lSlon,    lPlon,    lPpar,    lPper,
                    tLL, tUL,
                    sigma_offset, sigma_slope, sigma_curvature,
                    mu,
                    eta_bar_os, eta_bar_ss,
                    p0_os,  p1_os, p2_os,
                    p0_ss,  p1_ss, p2_ss,
                    dp0_os, dp1_os, dp2_os,
                    dp0_ss, dp1_ss, dp2_ss,
                    nknots, knots, coeffs,
                    angular_weights);

  __syncthreads();
  for(int k = 0; k < 10; k++)
  {{
    atomicAdd( &w[0]+k , w10[k]);
    //w[k] += w10[k];
  }}

}}

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// GLOBAL::getAngularWeights ///////////////////////////////////////////////////

__global__
void pyAngularCov(double *dtrue, double *dreco, double *weight, double w[10], double cov[10][10], double scale,
  // Time-dependent angular distribution
  double G, double DG, double DM,
  double * CSP,
  double *ASlon, double *APlon, double *APpar, double *APper,
  double  pSlon, double  pPlon, double  pPpar, double  pPper,
  double *dSlon, double  dPlon, double  dPpar, double  dPper,
  double  lSlon, double  lPlon, double  lPpar, double  lPper,
  double tLL, double tUL,
  double sigma_offset, double sigma_slope, double sigma_curvature,
  double mu,
  // Flavor tagging
  double eta_bar_os, double eta_bar_ss,
  double p0_os,  double p1_os, double p2_os,
  double p0_ss,  double p1_ss, double p2_ss,
  double dp0_os, double dp1_os, double dp2_os,
  double dp0_ss, double dp1_ss, double dp2_ss,
  // Time acceptance
  int nknots, double *knots, double *coeffs,
  // Angular acceptance
  double *angular_weights,
                  int Nevt)
{{
  int evt = threadIdx.x + blockDim.x * blockIdx.x;
  if (evt >= Nevt) {{ return; }}

  if ( (DEBUG > 0) && ( threadIdx.x + blockDim.x * blockIdx.x == 0) )
  {{
    printf("\n====================================================================================================");
    printf("\nDEBUGGING IS ENABLED AND RUNNING\n");
    printf("====================================================================================================\n");
  }}

  double w10[10]       = {{0.0}};
  double cov10[10][10] = {{{{0.0}}}};
  double vec_true[9] = {{dtrue[evt*10+0], // cosK
                      dtrue[evt*10+1], // cosL
                      dtrue[evt*10+2], // hphi
                      dtrue[evt*10+3], // time
                      dtrue[evt*10+5], // sigma_t
                      dtrue[evt*10+6],  // qOS
                      dtrue[evt*10+6],  // qSS
                      0,  // etaOS
                      0  // etaSS
                    }};
  double vec_reco[9] = {{dreco[evt*10+0], // cosK
                      dreco[evt*10+1], // cosL
                      dreco[evt*10+2], // hphi
                      dreco[evt*10+3], // time
                      dreco[evt*10+5], // sigma_t
                      dreco[evt*10+6],  // qOS
                      dreco[evt*10+6],  // qSS
                      0,  // etaOS
                      0  // etaSS
                    }};
  //double scale = 3554770.373949724;
  getAngularWeights(vec_true, vec_reco, 1, w10,
    G, DG, DM, CSP[0],
    ASlon[0], APlon[0], APpar[0], APper[0],
    pSlon,    pPlon,    pPpar,    pPper,
    dSlon[0], dPlon,    dPpar,    dPper,
    lSlon,    lPlon,    lPpar,    lPper,
    tLL, tUL,
    sigma_offset, sigma_slope, sigma_curvature,
    mu,
    eta_bar_os, eta_bar_ss,
    p0_os,  p1_os, p2_os,
    p0_ss,  p1_ss, p2_ss,
    dp0_os, dp1_os, dp2_os,
    dp0_ss, dp1_ss, dp2_ss,
    nknots, knots, coeffs,
    angular_weights);

  // __syncthreads();
  // for(int k = 0; k < 10; k++)
  // {{
  //   atomicAdd( &w[0]+k , w10[k]);
  // }}
  // __syncthreads();

  if (DEBUG > 0 && ( threadIdx.x + blockDim.x * blockIdx.x < 3) )
  {{
    printf("\n");
    printf("w10 = %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E\n",
           w10[0],w10[1],w10[2],w10[3],w10[4],
           w10[5],w10[6],w10[7],w10[8],w10[9]);
    printf("w  = %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E\n",
           w[0]/scale,w[1]/scale,w[2]/scale,w[3]/scale,w[4]/scale,
           w[5]/scale,w[6]/scale,w[7]/scale,w[8]/scale,w[9]/scale);
    printf("\n");
  }}

  for(int i=0; i<10; i++)
  {{
    for(int j=i; j<10; j++)
    {{
      cov10[i][j] = (w10[i]-w[i]/scale)*(w10[j]-w[j]/scale)*weight[evt]*weight[evt];
    }}
  }}

  if (DEBUG > 0 && ( threadIdx.x + blockDim.x * blockIdx.x == 0) )
  {{
    printf("COV 0\n");
  }}
  for(int i=0; i<10; i++)
  {{
    if (DEBUG > 0 && ( threadIdx.x + blockDim.x * blockIdx.x == 0) )
    {{
      printf("%+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E\n",
             cov10[i][0],cov10[i][1],cov10[i][2],cov10[i][3],cov10[i][4],
             cov10[i][5],cov10[i][6],cov10[i][7],cov10[i][8],cov10[i][9]);
    }}
  }}
  if (DEBUG > 0 && ( threadIdx.x + blockDim.x * blockIdx.x == 0) )
  {{
    printf("COV 1\n");
  }}
  for(int i=0; i<10; i++)
  {{
    if (DEBUG > 0 && ( threadIdx.x + blockDim.x * blockIdx.x == 1) )
    {{
      printf("%+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E\n",
             cov10[i][0],cov10[i][1],cov10[i][2],cov10[i][3],cov10[i][4],
             cov10[i][5],cov10[i][6],cov10[i][7],cov10[i][8],cov10[i][9]);
    }}
  }}
  if (DEBUG > 0 && ( threadIdx.x + blockDim.x * blockIdx.x == 0) )
  {{
    printf("COV 2\n");
  }}
  for(int i=0; i<10; i++)
  {{
    if (DEBUG > 0 && ( threadIdx.x + blockDim.x * blockIdx.x == 2) )
    {{
      printf("%+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E\n",
             cov10[i][0],cov10[i][1],cov10[i][2],cov10[i][3],cov10[i][4],
             cov10[i][5],cov10[i][6],cov10[i][7],cov10[i][8],cov10[i][9]);
    }}
  }}
  if (DEBUG > 0 && ( threadIdx.x + blockDim.x * blockIdx.x == 0) )
  {{
    printf("\n");
  }}

  // if (DEBUG > 3 && ( threadIdx.x + blockDim.x * blockIdx.x == DEBUG_EVT) )
  // {{
  //   printf("\n");
  //   printf("%+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g\n",
  //          w[0],w[1],w[2],w[3],w[4],
  //          w[5],w[6],w[7],w[8],w[9]);
  //   printf("%+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g, %+1.5g\n",
  //          w10[0],w10[1],w10[2],w10[3],w10[4],
  //          w10[5],w10[6],w10[7],w10[8],w10[9]);
  //   printf("\n");
  // }}

  __syncthreads();
  for(int i=0; i<10; i++)
  {{
    for(int j=0; j<10; j++)
    {{
      atomicAdd( &cov[i][j], cov10[i][j] );
      //cov10[i][j] = (w10[i]-w[i])*(w10[j]-w[j]);
    }}
    if (DEBUG > 0 && ( threadIdx.x + blockDim.x * blockIdx.x == DEBUG_EVT) )
    {{
      printf("%+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E  %+.5E\n",
             cov[i][0],cov[i][1],cov[i][2],cov[i][3],cov[i][4],
             cov[i][5],cov[i][6],cov[i][7],cov[i][8],cov[i][9]);
    }}
  }}

}}

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// Acceptance //////////////////////////////////////////////////////////////////

__global__
void pySingleTimeAcc(double *time, double *lkhd, double *coeffs,
                     double mu, double sigma, double gamma,
                     double tLL, double tUL, int Nevt)
/*
This is a pycuda iterating function. It calls getAcceptanceSingle for each event
in time and returns
*/
{{

  int row = threadIdx.x + blockDim.x * blockIdx.x;
  if (row >= Nevt) {{ return; }}
  double t = time[row] - mu;
  // if (row==0){{
  //   printf("mu = %lf, sigma = %lf, gamma = %lf, tLL = %lf, tUL= %lf\n", mu, sigma, gamma, tLL, tUL);
  // }}
  lkhd[row] = getOneSplineTimeAcc(t, coeffs, sigma, gamma, tLL, tUL);

}}



__global__
void pyRatioTimeAcc(double *time1, double *time2,
                    double *lkhd1, double *lkhd2,
                    double *c1, double *c2,
                    double mu1, double sigma1, double gamma1,
                    double mu2, double sigma2, double gamma2,
                    double tLL, double tUL,
                    int Nevt1, int Nevt2)
/*
This is a pycuda iterating function. It calls getAcceptanceSingle for each event
in time and returns
*/
{{

  int row = threadIdx.x + blockDim.x * blockIdx.x;
  if (row < Nevt1)
  {{
    double t1 = time1[row] - mu1;
    lkhd1[row] = getOneSplineTimeAcc(t1, c1,     sigma1, gamma1, tLL, tUL);
  }}
  if (row < Nevt2)
  {{
    double t2 = time2[row] - mu2;
    lkhd2[row] = getTwoSplineTimeAcc(t2, c1, c2, sigma2, gamma2, tLL, tUL);
  }}
}}



__global__
void pyFullTimeAcc(double *time1, double *time2, double *time3,
                    double *lkhd1, double *lkhd2, double *lkhd3,
                    double *c1, double *c2, double *c3,
                    double mu1, double sigma1, double gamma1,
                    double mu2, double sigma2, double gamma2,
                    double mu3, double sigma3, double gamma3,
                    double tLL, double tUL,
                    int Nevt1, int Nevt2, int Nevt3)
/*
This is a pycuda iterating function. It calls getAcceptanceSingle for each event
in time and returns
*/
{{

  int row1 = threadIdx.x + blockDim.x * blockIdx.x;
  //int row2 = threadIdx.y + blockDim.y * blockIdx.y;
  //int row3 = threadIdx.z + blockDim.z * blockIdx.z;
  if (row1 < Nevt1)
  {{
    double t1 = time1[row1] - mu1;
    lkhd1[row1] = getOneSplineTimeAcc(t1, c1,     sigma1, gamma1, tLL, tUL);
  }}
  if (row1 < Nevt2)
  {{
    double t2 = time2[row1] - mu2;
    lkhd2[row1] = getTwoSplineTimeAcc(t2, c1, c2, sigma2, gamma2, tLL, tUL);
  }}
  if (row1 < Nevt3)
  {{
    double t3 = time3[row1] - mu3;
    lkhd3[row1] = getTwoSplineTimeAcc(t3, c2, c3, sigma3, gamma3, tLL, tUL);
  }}

}}
