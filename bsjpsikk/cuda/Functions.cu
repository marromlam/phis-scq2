#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//                                                                            //
//                       CUDA decay rate Bs -> mumuKK                         //
//                                                                            //
//   Created: 2019-01-25                                                      //
//  Modified: 2019-11-21                                                      //
//    Author: Marcos Romero                                                   //
//                                                                            //
//    This file is part of phis-scq packages, Santiago's framework for the    //
//                     phi_s analysis in Bs -> Jpsi K+ K-                     //
//                                                                            //
//  This file contains the following __kernels:                               //
//    * pyDiffRate: Computes Bs2MuMuKK pdf looping over the events. Now it    //
//                  handles a binned X_M fit without splitting beforehand the //
//                  data --it launches a thread per mass bin.                 //
//                                                                            //
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// Include headers /////////////////////////////////////////////////////////////

#include <stdio.h>
#include <math.h>
#include <pycuda-complex.hpp>

#define ERRF_CONST 1.12837916709551
#define XLIM 5.33
#define YLIM 4.29

////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// Functions ///////////////////////////////////////////////////////////////////



__device__ double factorial(int n)
{
   if(n <= 0)
    return 1.;

   double x = 1;
   int b = 0;
   do {
      b++;
      x *= b;
   } while(b!=n);

   return x;
}



__device__
double getTimeCal(double sigma,
                  double sigma_offset, double sigma_slope, double sigma_curvature)
{
  return sigma_curvature*sigma*sigma + sigma_slope*sigma + sigma_offset;
}



__device__ pycuda::complex<double> faddeeva(pycuda::complex<double> z)//, double t)
{
   double in_real = pycuda::real(z);
   double in_imag = pycuda::imag(z);
   int n, nc, nu;
   double h, q, Saux, Sx, Sy, Tn, Tx, Ty, Wx, Wy, xh, xl, x, yh, y;
   double Rx [33];
   double Ry [33];

   x = fabs(in_real);
   y = fabs(in_imag);

   if (y < YLIM && x < XLIM) {
      q = (1.0 - y / YLIM) * sqrt(1.0 - (x / XLIM) * (x / XLIM));
      h  = 1.0 / (3.2 * q);
      nc = 7 + int(23.0 * q);
//       xl = pow(h, double(1 - nc));
      double h_inv = 1./h;
      xl = h_inv;
      for(int i = 1; i < nc-1; i++)
          xl *= h_inv;

      xh = y + 0.5 / h;
      yh = x;
      nu = 10 + int(21.0 * q);
      Rx[nu] = 0.;
      Ry[nu] = 0.;
      for (n = nu; n > 0; n--){
         Tx = xh + n * Rx[n];
         Ty = yh - n * Ry[n];
         Tn = Tx*Tx + Ty*Ty;
         Rx[n-1] = 0.5 * Tx / Tn;
         Ry[n-1] = 0.5 * Ty / Tn;
         }
      Sx = 0.;
      Sy = 0.;
      for (n = nc; n>0; n--){
         Saux = Sx + xl;
         Sx = Rx[n-1] * Saux - Ry[n-1] * Sy;
         Sy = Rx[n-1] * Sy + Ry[n-1] * Saux;
         xl = h * xl;
      };
      Wx = ERRF_CONST * Sx;
      Wy = ERRF_CONST * Sy;
   }
   else {
      xh = y;
      yh = x;
      Rx[0] = 0.;
      Ry[0] = 0.;
      for (n = 9; n>0; n--){
         Tx = xh + n * Rx[0];
         Ty = yh - n * Ry[0];
         Tn = Tx * Tx + Ty * Ty;
         Rx[0] = 0.5 * Tx / Tn;
         Ry[0] = 0.5 * Ty / Tn;
      };
      Wx = ERRF_CONST * Rx[0];
      Wy = ERRF_CONST * Ry[0];
   }

   if (y == 0.) {
      Wx = exp(-x * x);
   }
   if (in_imag < 0.) {

      double exp_x2_y2 = exp(y * y - x * x);
      Wx =   2.0 * exp_x2_y2 * cos(2.0 * x * y) - Wx;
      Wy = - 2.0 * exp_x2_y2 * sin(2.0 * x * y) - Wy;
      if (in_real > 0.) {
         Wy = -Wy;
      }
   }
   else if (in_real < 0.) {
      Wy = -Wy;
   }

   return pycuda::complex<double>(Wx,Wy);
}



////////////////////////////////////////////////////////////////////////////////
