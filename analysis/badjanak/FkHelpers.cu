#include "hip/hip_runtime.h"
//this methods are helpful for the phi integration
//calculates int x^n * sin(x) dx
#include <ipanema/complex.hpp>

#ifdef CUDA
  WITHIN_KERNEL
  int convert_float(float in) {
      union fi { int i; float f; } conv;
      conv.f = in;
      return conv.i;
  }
#endif


WITHIN_KERNEL ftype integral_x_to_n_times_sin_x(ftype x, int n);
WITHIN_KERNEL ftype integral_x_to_n_times_cos_x(ftype x, int n);
WITHIN_KERNEL ftype integral_x_to_n_times_sin_2x(ftype x, int n);
WITHIN_KERNEL ftype integral_x_to_n_times_cos_2x(ftype x, int n);
WITHIN_KERNEL ftype integral_x_to_n_times_sqrt_1_minus_x2(ftype x, int n);

//this methods are helpful for the phi integration
//calculates int x^n * sin(x) dx
WITHIN_KERNEL
ftype integral_x_to_n_times_sin_x(ftype x, int n)
{
  //ftype N = convert_float(n);
  if (n == 0)
    return -cos(x);
  else
    return -pow(x, n)*cos(x) + n*integral_x_to_n_times_cos_x(x, n-1);
}

//calculates int x^n * cos(x) dx
WITHIN_KERNEL
ftype integral_x_to_n_times_cos_x(ftype x, int n)
{
  //ftype N = convert_float(n);
  if (n == 0)
    return sin(x);
  else
    return pow(x,n)*sin(x) - n*integral_x_to_n_times_sin_x(x, n-1);
}

//calculates int x^n * sin(2x) dx
WITHIN_KERNEL
ftype integral_x_to_n_times_sin_2x(ftype x, int n)
{
  //ftype N = convert_float(n);
  if (n == 0)
    return -0.5*cos(2.0*x);
  else
    return -pow(x,n)*0.5*cos(2.0*x)
      +0.5*n*integral_x_to_n_times_cos_2x(x,n-1);
}

//calculates int x^n * cos(2x) dx
WITHIN_KERNEL
ftype integral_x_to_n_times_cos_2x(ftype x, int n)
{
  //ftype N = convert_float(n);
  if (n == 0)
    return 0.5*sin(2.0*x);
  else
    return +0.5*pow(x,n)*sin(2.0*x)
      -0.5*n*integral_x_to_n_times_sin_2x(x,n-1);
}

//calculates int x^n * cos(x)^2 dx
WITHIN_KERNEL
ftype integral_x_to_n_times_cos_x_2(ftype x, int n)
{
  //ftype N = convert_float(n);
  return +1.0/(1.0 + n)*pow(x,n+1)*cos(x)*cos(x)
    +1.0/(1.0+n)*integral_x_to_n_times_sin_2x(x, n+1);
}

//calculates int x^n * sin(x)^2 dx
WITHIN_KERNEL
ftype integral_x_to_n_times_sin_x_2(ftype x, int n)
{
  //ftype N = convert_float(n);
  return +1.0/(1.0 + n)*pow(x,n+1)*sin(x)*sin(x)
    -1.0/(1.0+n)*integral_x_to_n_times_sin_2x(x, n+1);
}

//calculates int x^n * asin(x) dx
WITHIN_KERNEL
ftype integral_x_to_n_times_asin_x(ftype x, int n)
{
  //ftype N = convert_float(n);
  if (n == 0)
    return x*asin(x)+sqrt(1-x*x);
  else
    return 1.0/(n+1.0)*pow(x,n)*(x*asin(x)+sqrt(1-x*x))
      -n*integral_x_to_n_times_sqrt_1_minus_x2(x, n-1);
}

//calculates int x^n * sqrt(1-x^2) dx
WITHIN_KERNEL
ftype integral_x_to_n_times_sqrt_1_minus_x2(ftype x, int n)
{
  //ftype N = convert_float(n);
  if (n == 0)
    return 0.5*asin(x)+0.5*x*sqrt(1-x*x);
  else
    return 2.0/(n+2.0)*pow(x, n)*(0.5*asin(x)+0.5*x*sqrt(1-x*x))
      -n/(n+2.0)*integral_x_to_n_times_asin_x(x, n-1);
}




WITHIN_KERNEL
ftype integral_ijk_f1(ftype cosKa, ftype cosKb,
                                      ftype cosLa, ftype cosLb,
                                      ftype phia, ftype phib,
                                      int k, int i, int j)
{
  //assert(i>=0);
  //assert(j>=0);
  //assert(k>=0);
  ftype c0 = 9.0/(32.0*M_PI);
  return
    2.0*c0
    *(pow(cosKb,k+3)-pow(cosKa,k+3))/(k+3) //cosK
    *(pow(cosLb,i+1)/(i+1)-pow(cosLb,i+3)/(i+3)-pow(cosLa,i+1)/(i+1)+pow(cosLa,i+3)/(i+3)) //cosL
    *(pow(phib,j+1)-pow(phia,j+1))/(j+1); //phi
}



WITHIN_KERNEL
ftype integral_ijk_f2(ftype cosKa, ftype cosKb,
                                      ftype cosLa, ftype cosLb,
                                      ftype phia, ftype phib,
                                      int k, int i, int j)
{
  //assert(i>=0);
  //assert(j>=0);
  //assert(k>=0);
  ftype c0 = 9.0/(32.0*M_PI);
  return
   c0
   *(pow(cosKb,k+1)/(k+1)-pow(cosKb,k+3)/(k+3)-pow(cosKa,k+1)/(k+1)+pow(cosKa,k+3)/(k+3)) //cosK
   *(
     (pow(phib,j+1)-pow(phia,j+1))/(j+1)//phi1
     *(pow(cosLb,i+1)-pow(cosLa,i+1))/(i+1)//cosK1
     -(pow(cosLb,i+1)/(i+1)-pow(cosLb,i+3)/(i+3)-pow(cosLa,i+1)/(i+1)+pow(cosLa,i+3)/(i+3)) //cosK2
     *(integral_x_to_n_times_cos_x_2(phib, j)-integral_x_to_n_times_cos_x_2(phia, j))//phi2
     );
}


WITHIN_KERNEL
ftype integral_ijk_f3(ftype cosKa, ftype cosKb,
                                      ftype cosLa, ftype cosLb,
                                      ftype phia, ftype phib,
                                      int k, int i, int j)
{
  //assert(i>=0);
  //assert(j>=0);
  //assert(k>=0);
  ftype c0 = 9.0/(32.0*M_PI);
  return
    c0
    *(pow(cosKb,k+1)/(k+1)-pow(cosKb,k+3)/(k+3)-pow(cosKa,k+1)/(k+1)+pow(cosKa,k+3)/(k+3)) //cosL
    *(
      (pow(phib,j+1)-pow(phia,j+1))/(j+1)//phi1
      *(pow(cosLb,i+1)-pow(cosLa,i+1))/(i+1)//cosK1
      -(pow(cosLb,i+1)/(i+1)-pow(cosLb,i+3)/(i+3)-pow(cosLa,i+1)/(i+1)+pow(cosLa,i+3)/(i+3)) //cosK2
      *(integral_x_to_n_times_sin_x_2(phib, j)-integral_x_to_n_times_sin_x_2(phia, j))//phi2
      );
}


WITHIN_KERNEL
ftype integral_ijk_f4(ftype cosKa, ftype cosKb,
                                      ftype cosLa, ftype cosLb,
                                      ftype phia, ftype phib,
                                      int k, int i, int j)
{
  //assert(i>=0);
  //assert(j>=0);
  //assert(k>=0);
  ftype c0 = 9.0/(32.0*M_PI);
  return
    c0
    *(pow(cosKb,k+1)/(k+1)-pow(cosKb,k+3)/(k+3)-pow(cosKa,k+1)/(k+1)+pow(cosKa,k+3)/(k+3)) //cosL
    *(pow(cosLb,i+1)/(i+1)-pow(cosLb,i+3)/(i+3)-pow(cosLa,i+1)/(i+1)+pow(cosLa,i+3)/(i+3)) //cosK
    *(integral_x_to_n_times_sin_2x(phib, j) - integral_x_to_n_times_sin_2x(phia, j));//phi
}


WITHIN_KERNEL
ftype integral_ijk_f5(ftype cosKa, ftype cosKb,
                                      ftype cosLa, ftype cosLb,
                                      ftype phia, ftype phib,
                                      int k, int i, int j)
{
  //assert(i>=0);
  //assert(j>=0);
  //assert(k>=0);
  ftype c0 = 9.0/(32.0*M_PI);
  return
    c0/sqrt(2.0)
    *2.0*(integral_x_to_n_times_sqrt_1_minus_x2(cosKb, k+1) - integral_x_to_n_times_sqrt_1_minus_x2(cosKa, k+1))//cosL
    *2.0*(integral_x_to_n_times_sqrt_1_minus_x2(cosLb, i+1) - integral_x_to_n_times_sqrt_1_minus_x2(cosLa, i+1))//cosK
    *(integral_x_to_n_times_cos_x(phib, j) - integral_x_to_n_times_cos_x(phia, j));//phi
}


WITHIN_KERNEL
ftype integral_ijk_f6(ftype cosKa, ftype cosKb,
                                      ftype cosLa, ftype cosLb,
                                      ftype phia, ftype phib,
                                      int k, int i, int j)
{
  //assert(i>=0);
  //assert(j>=0);
  //assert(k>=0);
  ftype c0 = 9.0/(32.0*M_PI);
  return
    -c0/sqrt(2.0)
    *2.0*(integral_x_to_n_times_sqrt_1_minus_x2(cosKb, k+1) - integral_x_to_n_times_sqrt_1_minus_x2(cosKa, k+1))//cosL
    *2.0*(integral_x_to_n_times_sqrt_1_minus_x2(cosLb, i+1) - integral_x_to_n_times_sqrt_1_minus_x2(cosLa, i+1))//cosK
    *(integral_x_to_n_times_sin_x(phib, j) - integral_x_to_n_times_sin_x(phia, j));//phi
}


WITHIN_KERNEL
ftype integral_ijk_f7(ftype cosKa, ftype cosKb,
                                      ftype cosLa, ftype cosLb,
                                      ftype phia, ftype phib,
                                      int k, int i, int j)
{
  //assert(i>=0);
  //assert(j>=0);
  //assert(k>=0);
  const ftype c0 = +3.0/(32.0*M_PI);
  return
    c0*2.0
    *(pow(cosLb,i+1)/(i+1)-pow(cosLb,i+3)/(i+3)-pow(cosLa,i+1)/(i+1)+pow(cosLa,i+3)/(i+3)) //cosK
    *(pow(phib,j+1)-pow(phia,j+1))/(j+1) //phi
    *(pow(cosKb,k+1)-pow(cosKa,k+1))/(k+1); //cosL
}


WITHIN_KERNEL
ftype integral_ijk_f8(ftype cosKa, ftype cosKb,
                                      ftype cosLa, ftype cosLb,
                                      ftype phia, ftype phib,
                                      int k, int i, int j)
{
  const ftype c0 = +3.0/(32.0*M_PI);
  return
    c0*sqrt(6.0)
    *(integral_x_to_n_times_sqrt_1_minus_x2(cosKb, k)-integral_x_to_n_times_sqrt_1_minus_x2(cosKa, k))//cosL
    *(integral_x_to_n_times_cos_x(phib, j) - integral_x_to_n_times_cos_x(phia, j))//phi
    *2.0*(integral_x_to_n_times_sqrt_1_minus_x2(cosLb, i+1) - integral_x_to_n_times_sqrt_1_minus_x2(cosLa, i+1));//cosK
}


WITHIN_KERNEL
ftype integral_ijk_f9(ftype cosKa, ftype cosKb,
                                      ftype cosLa, ftype cosLb,
                                      ftype phia, ftype phib,
                                      int k, int i, int j)
{
  const ftype c0 = +3.0/(32.0*M_PI);
  return
    -c0*sqrt(6.0)
    *(integral_x_to_n_times_sqrt_1_minus_x2(cosKb, k)-integral_x_to_n_times_sqrt_1_minus_x2(cosKa, k))//cosL
    *(integral_x_to_n_times_sin_x(phib, j) - integral_x_to_n_times_sin_x(phia, j))//phi
    *2.0*(integral_x_to_n_times_sqrt_1_minus_x2(cosLb, i+1) - integral_x_to_n_times_sqrt_1_minus_x2(cosLa, i+1));//cosK
}


WITHIN_KERNEL
ftype integral_ijk_f10(ftype cosKa, ftype cosKb,
                                       ftype cosLa, ftype cosLb,
                                       ftype phia, ftype phib,
                                       int k, int i, int j)
{
  const ftype c0 = +3.0/(32.0*M_PI);
  return
    c0*4.0*sqrt(3.0)
    *(pow(cosKb,k+2)-pow(cosKa,k+2))/(k+2) //cosL
    *(pow(phib,j+1)-pow(phia,j+1))/(j+1) //phi
    *(pow(cosLb,i+1)/(i+1)-pow(cosLb,i+3)/(i+3)-pow(cosLa,i+1)/(i+1)+pow(cosLa,i+3)/(i+3)); //cosK
}



KERNEL
void integral_ijk_fx(ftype cosKs, ftype cosKe, ftype cosLs, ftype cosLe, 
                     ftype phis, ftype phie, int i, int j, int k,
                     GLOBAL_MEM ftype * fx)
{
  fx[0] = integral_ijk_f1( cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
  fx[1] = integral_ijk_f2( cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
  fx[2] = integral_ijk_f3( cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
  fx[3] = integral_ijk_f4( cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
  fx[4] = integral_ijk_f5( cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
  fx[5] = integral_ijk_f6( cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
  fx[6] = integral_ijk_f7( cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
  fx[7] = integral_ijk_f8( cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
  fx[8] = integral_ijk_f9( cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
  fx[9] = integral_ijk_f10(cosKs, cosKe, cosLs, cosLe, phis, phie, i, j, k);
}







WITHIN_KERNEL
ftype ang_eff(const ftype cosK, const ftype cosL, const ftype phi, ftype *moments)
{
    ftype eff = 0.;

    eff += moments[0] * legendre_poly(0, 0, cosK) * sph_harm(0, 0, cosL, phi);
    eff += moments[1] * legendre_poly(0, 0, cosK) * sph_harm(2, 0, cosL, phi);
    eff += moments[2] * legendre_poly(0, 0, cosK) * sph_harm(2, 2, cosL, phi);
    eff += moments[3] * legendre_poly(0, 0, cosK) * sph_harm(2, 1, cosL, phi);
    eff += moments[4] * legendre_poly(0, 0, cosK) * sph_harm(2,-1, cosL, phi);
    eff += moments[5] * legendre_poly(0, 0, cosK) * sph_harm(2,-2, cosL, phi);
    eff += moments[6] * legendre_poly(1, 0, cosK) * sph_harm(0, 0, cosL, phi);
    eff += moments[7] * legendre_poly(1, 0, cosK) * sph_harm(2, 1, cosL, phi);
    eff += moments[8] * legendre_poly(1, 0, cosK) * sph_harm(2,-1, cosL, phi);
    eff += moments[9] * legendre_poly(2, 0, cosK) * sph_harm(0, 0, cosL, phi);

    eff *= 2.*sqrt(M_PI);
//     printf("Ang. eff = %lf \n", eff);
    return eff;
}




WITHIN_KERNEL
void angWeightsToMoments(ftype* moments, GLOBAL_MEM const ftype* normweights)
{
    //c0000
    moments[0]  =   1. / 3. * ( normweights[0] + normweights[1] + normweights[2] );//
    //c0020
    moments[1]  =   1. / 3. * sqrt(5.)             * ( normweights[0] + normweights[1] + normweights[2] - 3. * normweights[6] );//
    //c0022
    moments[2]  =            -sqrt(5. / 3.)        * ( normweights[1] - normweights[2] );//
    //c0021
    moments[3]  = - 8. / 3. * sqrt( 5. / 2. ) / M_PI *   normweights[7];//
    //c002-1
    moments[4]  = - 8. / 3. * sqrt( 5. / 2. ) / M_PI *  (normweights[8]);//-normweights should be +normweights?
    //c002-2
    moments[5]  =             sqrt( 5. / 3. )      *  (normweights[3]);//-normweights should be +normweights?
    //c1000
    moments[6]  =   1. / 2. * sqrt(3.)             *   normweights[9];//
    //c1021
    moments[7]  = - 32. / 3. * sqrt( 5. / 6. ) / M_PI *   normweights[4];//
    //c102-1
    moments[8]  = + 32. / 3. * sqrt( 5. / 6. ) / M_PI *  (normweights[5]);//-normweights should be +normweights?
    //c2000
    moments[9]  =  5. / 2.                        * ( normweights[0] - normweights[6] );//
}






KERNEL
void plot_moments(GLOBAL_MEM const ftype *normweights, GLOBAL_MEM ftype *out,
                  GLOBAL_MEM const ftype *cosK, GLOBAL_MEM const ftype *cosL, 
                  GLOBAL_MEM const ftype *hphi)
{
  const int i = get_global_id(0);

  ftype moments[10] = {0.,0.,0.,0.,0.,0.,0.,0.,0.,0.};

  // get the moments
  angWeightsToMoments(moments, normweights);
  out[i] = ang_eff(cosK[i], cosL[i], hphi[i], moments);

  //ftype ang_acc = ang_eff(x, y, z, moments); // these are angular weights again

}
