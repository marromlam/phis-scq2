#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//                                                                            //
//                         DIFFERENTIAL CROSS RATE                            //
//                                                                            //
//   Created: 2020-06-25                                                      //
//    Author: Marcos Romero Lamas (mromerol@cern.ch)                          //
//                                                                            //
//    This file is part of phis-scq packages, Santiago's framework for the    //
//                     phi_s analysis in Bs -> Jpsi K+ K-                     //
//                                                                            //
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////











////////////////////////////////////////////////////////////////////////////////
// Functions ///////////////////////////////////////////////////////////////////

//Ramon es mejor pasar el data igual para no confundirse en otras partes del codigo

WITHIN_KERNEL
ftype getDiffRateBd(const ftype *data,
                    const ftype CSP,
                    const ftype ASlon, const ftype APlon, const ftype APpar, 
                    const ftype APper, const ftype dSlon, const ftype dPlon, 
                    const ftype dPpar, const ftype dPper,
                    // Angular acceptance
                    GLOBAL_MEM  const ftype *angular_weights,
                    const int USE_FK
                  )
{


  // Variables -----------------------------------------------------------------
  //     Make sure that the input it's in this order.
  //     lalala
  ftype cosK       = data[0];                      // Time-angular distribution
  ftype cosL       = data[1];
  ftype hphi       = data[2];                            // Time resolution
  ftype qOS        = data[5];                                      // Tagging



  // Flavor tagging ------------------------------------------------------------
  ftype id;
  id = qOS/fabs(qOS);

  // Compute per event pdf -----------------------------------------------------
  ftype fk, ak;
  ftype pdfB = 0.0;
  ftype norm = 0.0;

  for(int k = 1; k <= 10; k++)
  {
    if (USE_FK)
    {
      fk = ( 9.0/(16.0*M_PI) )*getF(cosK,cosL,hphi,k);
    }
    else
    {
      fk = TRISTAN[k-1];
    }
    ak = getAbd(ASlon, APlon, APpar, APper, dSlon, dPpar, dPpar, dPper, CSP, k);
    norm += angular_weights[k-1]*ak;
    //norm += TRISTAN[k-1]*ak;
    if ( (k==4) || (k==6)  || (k==9) )
    {
      pdfB += id*fk*ak;
    }
    else
    {
      pdfB += fk*ak;
    }
  }


  return pdfB/norm;
}
